
#include "tiger/utils/math_function.hpp"
#include "tiger/common.hpp"
#include "tiger/utils/device_alternate.hpp"

namespace tiger{
void tiger_gpu_memcpy(const size_t N, const void* src, void* des){
    if(src != des){
	CUDA_CHECK(hipMemcpy(des, src, N, hipMemcpyDefault));
    }
}

template <>
void tiger_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
	const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
	const float alpha, const float* A, const float* B, const float beta,
	float* C){
    
    int lda = (TransA == CblasNoTrans) ? K : M;
    int ldb = (TransB == CblasNoTrans) ? N : K;
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB = 
	(TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    // hipblasSgemm是float的的矩阵乘法
    hipblasHandle_t cublas_handle; 
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasSgemm(cublas_handle, cuTransB, cuTransA,
		N, M, K, &alpha, B, ldb, A, lda, &beta, C, N)); 
}

template <>
void tiger_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
	const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
	const double alpha, const double* A, const double* B, const double beta,
	double* C){
    
    int lda = (TransA == CblasNoTrans) ? K : M;
    int ldb = (TransB == CblasNoTrans) ? N : K;
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB = 
	(TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    // hipblasDgemm是double的的矩阵乘法
    hipblasHandle_t cublas_handle; 
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, cuTransB, cuTransA,
		N, M, K, &alpha, B, ldb, A, lda, &beta, C, N)); 
}

template <>
void tiger_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA,
	const int M, const int N, const float alpha, const float* A, const float* x,
	const float beta, float* y){
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasSgemv(cublas_handle, cuTransA, N, M, &alpha,
		A, N, x, 1, &beta, y, 1));

}
template <>
void tiger_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA,
	const int M, const int N, const double alpha, const double* A, const double* x,
	const double beta, double* y){
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasDgemv(cublas_handle, cuTransA, N, M, &alpha,
		A, N, x, 1, &beta, y, 1));
}

template <>
void tiger_gpu_axpy<float>(const int N, const float alpha, const float* X, float* Y){
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasSaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
}

template <>
void tiger_gpu_axpy<double>(const int N, const double alpha, const double* X, double* Y){
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
}
}

