#include "tiger/utils/math_function.hpp"
#include "tiger/common.hpp"

namespace tiger{
void tiger_gpu_memcpy(const size_t N, const void* src, void* des){
    if(src != des){
	CUDA_CHECK(hipMemcpy(des, src, N, hipMemcpyDefault));
    }
}

}

