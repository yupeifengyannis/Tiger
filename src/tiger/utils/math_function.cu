#include "hip/hip_runtime.h"

#include "tiger/utils/math_function.hpp"
#include "tiger/common.hpp"
#include "tiger/utils/device_alternate.hpp"

namespace tiger{
void tiger_gpu_memcpy(const size_t N, const void* src, void* des){
    if(src != des){
	CUDA_CHECK(hipMemcpy(des, src, N, hipMemcpyDefault));
    }
}

template <>
void tiger_gpu_gemm<float>(const CBLAS_TRANSPOSE TransA,
	const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
	const float alpha, const float* A, const float* B, const float beta,
	float* C){

    int lda = (TransA == CblasNoTrans) ? K : M;
    int ldb = (TransB == CblasNoTrans) ? N : K;
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB = 
	(TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    // hipblasSgemm是float的的矩阵乘法
    hipblasHandle_t cublas_handle; 
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasSgemm(cublas_handle, cuTransB, cuTransA,
		N, M, K, &alpha, B, ldb, A, lda, &beta, C, N)); 
}

template <>
void tiger_gpu_gemm<double>(const CBLAS_TRANSPOSE TransA,
	const CBLAS_TRANSPOSE TransB, const int M, const int N, const int K,
	const double alpha, const double* A, const double* B, const double beta,
	double* C){

    int lda = (TransA == CblasNoTrans) ? K : M;
    int ldb = (TransB == CblasNoTrans) ? N : K;
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    hipblasOperation_t cuTransB = 
	(TransB == CblasNoTrans) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
    // hipblasDgemm是double的的矩阵乘法
    hipblasHandle_t cublas_handle; 
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasDgemm(cublas_handle, cuTransB, cuTransA,
		N, M, K, &alpha, B, ldb, A, lda, &beta, C, N)); 
}

template <>
void tiger_gpu_gemv<float>(const CBLAS_TRANSPOSE TransA,
	const int M, const int N, const float alpha, const float* A, const float* x,
	const float beta, float* y){
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasSgemv(cublas_handle, cuTransA, N, M, &alpha,
		A, N, x, 1, &beta, y, 1));

}
template <>
void tiger_gpu_gemv<double>(const CBLAS_TRANSPOSE TransA,
	const int M, const int N, const double alpha, const double* A, const double* x,
	const double beta, double* y){
    hipblasOperation_t cuTransA = 
	(TransA == CblasNoTrans) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasDgemv(cublas_handle, cuTransA, N, M, &alpha,
		A, N, x, 1, &beta, y, 1));
}

template <>
void tiger_gpu_axpy<float>(const int N, const float alpha, const float* X, float* Y){
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasSaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
}

template <>
void tiger_gpu_axpy<double>(const int N, const double alpha, const double* X, double* Y){
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasDaxpy(cublas_handle, N, &alpha, X, 1, Y, 1));
}


void tiger_gpu_rng_uniform(const int n, unsigned int* r){
    hiprandGenerator_t curand_generator;
    hiprandCreateGenerator(&curand_generator, HIPRAND_RNG_PSEUDO_MTGP32);
    CURAND_CHECK(hiprandGenerate(curand_generator, r, n));
}



template <>
void tiger_gpu_scal<float>(const int N, const float alpha, float *X) {
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasSscal(cublas_handle, N, &alpha, X, 1));
}

template <>
void tiger_gpu_scal<double>(const int N, const double alpha, double *X) {
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);
    CUBLAS_CHECK(hipblasDscal(cublas_handle, N, &alpha, X, 1));
}

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
	y[i] += alpha;
    }
}

template <>
void tiger_gpu_add_scalar<float>(const int N, const float alpha, float* Y){
    add_scalar_kernel<float><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, alpha, Y);
}


template <>
void tiger_gpu_add_scalar<double>(const int N, const double alpha, double* Y){
    add_scalar_kernel<double><<<GET_BLOCKS(N), CUDA_NUM_THREADS>>>(N, alpha, Y);
}

template <>
void tiger_gpu_rng_uniform<float>(const int n, const float a, const float b,
float* r) {
    hiprandGenerator_t curand_generator;
    hiprandCreateGenerator(&curand_generator, HIPRAND_RNG_PSEUDO_MTGP32);
    CURAND_CHECK(hiprandGenerateUniform(curand_generator, r, n));
    const float range = b - a;
    if (range != static_cast<float>(1)) {
	tiger_gpu_scal(n, range, r);
    }
    if (a != static_cast<float>(0)) {
	tiger_gpu_add_scalar(n, a, r);
    }
}

template <>
void tiger_gpu_rng_uniform<double>(const int n, const double a, const double b,
double* r) {
    hiprandGenerator_t curand_generator;
    hiprandCreateGenerator(&curand_generator, HIPRAND_RNG_PSEUDO_MTGP32);
    CURAND_CHECK(hiprandGenerateUniformDouble(curand_generator, r, n));
    const double range = b - a;
    if (range != static_cast<double>(1)) {
	tiger_gpu_scal(n, range, r);
    }
    if (a != static_cast<double>(0)) {
	tiger_gpu_add_scalar(n, a, r);
    }
}



}

